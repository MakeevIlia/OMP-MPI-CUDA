#include "hip/hip_runtime.h"
#include "mpi.h"
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cmath> 
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

using namespace std;


#define Pi 3.141592653589793

struct Point {
	float x, y, z, value, oldvalue;
};

__device__
float uanalitical(float at, float Lx, float Ly, float Lz, float x, float y, float z, float t)
{
	return sin((x * Pi) / Lx) * sin((2 * y * Pi) / Ly) * sin((3 * z * Pi) / Lz) * cos(at * t);
}


float phi(float Lx, float Ly, float Lz, float x, float y, float z)
{
	return sin((x * Pi) / Lx) * sin((2 * y * Pi) / Ly) * sin((3 * z * Pi) / Lz);
}


void multi(int n, int& a1, int& a2, int& a3)
{
	a1 = 1;
	a2 = 1;
	a3 = 1;

	int k = 1;
	int div = 2;
	while (n > 1)
	{
		while (n % div == 0)
		{
			n = n / div;
			if (k % 3 == 1)
			{
				a1 *= div;
				k++;
			}
			else
				if (k % 3 == 2)
				{
					a2 *= div;
					k++;
				}
				else
					if (k % 3 == 0)
					{
						a3 *= div;
						k++;
					}
		}
		div++;
	}
}

__global__
void counterror(float* GPUError, int Nxl, int Nyl, int Nzl, Point* GCP, float tauh, float at, float Lx, float Ly, float Lz, float t)
{
	const int blocksize = 512; // Число нитей в блоке
	int thread = threadIdx.x; // Вычисление адреса нити
	float error = 0; // Максимум ошибки для этой нити
	float value;
	float x;
	float y;
	float z;
	Point current;
	int n = (Nxl + 2) * (Nyl + 2) * (Nzl + 2); // Общее число элементов в каждоми блоке

	for (int id = thread; id < n; id += blocksize) //Для каждой нити "шагаем" по всем блокам
	{
		// Так как в блоке хранятся не только значения самого блока, но и граничные значения, то координаты необходимо исправить, чтобы они лежали в диапазоне от 1 до (максимум по этой координате - 1)
		int k = int(thread / ((Nxl + 2) * (Nyl + 2)));
		int j = int((thread - k * (Nxl + 2) * (Nyl + 2)) / Nxl);
		int i = int(thread - k * (Nxl + 2) * (Nyl + 2) - (Nxl + 2) * j);

		k = 1 + k % (Nzl + 1);
		j = 1 + j % (Nyl + 1);
		i = 1 + i % (Nxl + 1);


		current = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i]; // Берем точку из схемы и считаем для нее максимум
		x = current.x;
		y = current.y;
		z = current.z;
		value = current.value;

		float diff = fabs(value - uanalitical(at, Lx, Ly, Lz, x, y, z, t));

		if (diff > error)
		{
			error = diff; //Подсчет максимальной ошибки для этой нити
		}
	}

	// Далее каждая нить запишет в свой адрес максимум, который она вычислила по всем блокам

	__shared__ float ErrorPerThread[blocksize];
	ErrorPerThread[thread] = error;
	__syncthreads();

	for (int i = blocksize / 2; i > 0; i /= 2)
	{
		if (thread < i)
		{
			if (ErrorPerThread[thread] > ErrorPerThread[thread + i])
			{
				ErrorPerThread[thread] = ErrorPerThread[thread];
			}
			else
			{
				ErrorPerThread[thread] = ErrorPerThread[thread + i];
			}
		}
		__syncthreads();
	}

	if (thread == 0)
	{
		*GPUError = ErrorPerThread[0];
	}
}

__global__
void Kernel1(int Nxl, int Nyl, int Nzl, Point* GCP, Point* GLS, Point* GRS, Point* GDS, Point* GUS, Point* GFS, Point* GBS, float tauh)
{
	int thread = (blockIdx.x * blockDim.x) + threadIdx.x;
	int k = int(thread / ((Nxl + 2) * (Nyl + 2)));
	int j = int((thread - k * (Nxl + 2) * (Nyl + 2)) / Nxl);
	int i = int(thread - k * (Nxl + 2) * (Nyl + 2) - (Nxl + 2) * j);
	Point current = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i];
	float value;
	value = current.value;
	//printf("Thread = %10d, i = %10d, j = %10d, k = %10d, value =  %3.10f \n", thread, i, j, k, value );
	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = 2 * value - current.oldvalue + tauh * (GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i - 1].value + GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i + 1].value + GCP[k * (Nxl + 2) * (Nyl + 2) + (j - 1) * (Nxl + 2) + i].value + GCP[k * (Nxl + 2) * (Nyl + 2) + (j + 1) * (Nxl + 2) + i].value + GCP[(k - 1) * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value + GCP[(k + 1) * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value - 6 * value);
	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
	j = 1;
	GFS[k * (Nxl + 2) + i].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GFS[k * (Nxl + 2) + i].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GFS[k * (Nxl + 2) + i].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GFS[k * (Nxl + 2) + i].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GFS[k * (Nxl + 2) + i].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
	j = Nyl;
	GBS[k * (Nxl + 2) + i].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GBS[k * (Nxl + 2) + i].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GBS[k * (Nxl + 2) + i].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GBS[k * (Nxl + 2) + i].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GBS[k * (Nxl + 2) + i].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
	j = int((thread - k * (Nxl + 2) * (Nyl + 2)) / Nxl);
	i = 1;
	GLS[k * (Nyl + 2) + j].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GLS[k * (Nyl + 2) + j].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GLS[k * (Nyl + 2) + j].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GLS[k * (Nyl + 2) + j].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GLS[k * (Nyl + 2) + j].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
	i = Nxl;
	GRS[k * (Nyl + 2) + j].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GRS[k * (Nyl + 2) + j].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GRS[k * (Nyl + 2) + j].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GRS[k * (Nyl + 2) + j].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GRS[k * (Nyl + 2) + j].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
	i = int(thread - k * (Nxl + 2) * (Nyl + 2) - Nxl * j);
	k = 1;
	GDS[j * (Nxl + 2) + i].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GDS[j * (Nxl + 2) + i].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GDS[j * (Nxl + 2) + i].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GDS[j * (Nxl + 2) + i].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GDS[j * (Nxl + 2) + i].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
	k = Nzl;
	GUS[j * (Nxl + 2) + i].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GUS[j * (Nxl + 2) + i].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GUS[j * (Nxl + 2) + i].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GUS[j * (Nxl + 2) + i].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GUS[j * (Nxl + 2) + i].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
}

__global__
void Kernel2(int Nxl, int Nyl, int Nzl, Point* GCP, Point* GLS, Point* GRS, Point* GDS, Point* GUS, Point* GFS, Point* GBS, Point* GLR, Point* GRR, Point* GDR, Point* GUR, Point* GFR, Point* GBR, float tauh)
{
	int thread = (blockIdx.x * blockDim.x) + threadIdx.x;
	int k = int(thread / ((Nxl + 2) * (Nyl + 2)));
	int j = int((thread - k * (Nxl + 2) * (Nyl + 2)) / (Nxl + 2));
	int i = int(thread - k * (Nxl + 2) * (Nyl + 2) - (Nxl + 2) * j);

	k = 0;

	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = GDR[j * (Nxl + 2) + i].oldvalue;
	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = GDR[j * (Nxl + 2) + i].value;


	k = Nzl + 1;

	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = GUR[j * (Nxl + 2) + i].oldvalue;
	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = GUR[j * (Nxl + 2) + i].value;
	k = int(thread / ((Nxl + 2) * (Nyl + 2)));

	i = Nxl + 1;

	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = GRR[k * (Nyl + 2) + j].oldvalue;
	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = GRR[k * (Nyl + 2) + j].value;

	i = 0;

	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = GLR[k * (Nyl + 2) + j].oldvalue;
	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = GLR[k * (Nyl + 2) + j].value;
	i = int(thread - k * (Nxl + 2) * (Nyl + 2) - Nxl * j);

	j = Nyl + 1;

	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = GBR[k * (Nxl + 2) + i].oldvalue;
	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = GBR[k * (Nxl + 2) + i].value;


	j = 0;

	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = GFR[k * (Nxl + 2) + i].oldvalue;
	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = GFR[k * (Nxl + 2) + i].value;
	j = int((thread - k * (Nxl + 2) * (Nyl + 2)) / Nxl);


	Point current = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i];
	float value;
	value = current.value;
	//printf("Thread = %10d, i = %10d, j = %10d, k = %10d, value =  %3.10f \n", thread, i, j, k, value );
	__syncthreads();
	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = 2 * value - current.oldvalue + tauh * (GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i - 1].value + GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i + 1].value + GCP[k * (Nxl + 2) * (Nyl + 2) + (j - 1) * (Nxl + 2) + i].value + GCP[k * (Nxl + 2) * (Nyl + 2) + (j + 1) * (Nxl + 2) + i].value + GCP[(k - 1) * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value + GCP[(k + 1) * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value - 6 * value);
	GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;

	j = 1;
	GFS[k * (Nxl + 2) + i].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GFS[k * (Nxl + 2) + i].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GFS[k * (Nxl + 2) + i].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GFS[k * (Nxl + 2) + i].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GFS[k * (Nxl + 2) + i].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
	j = Nyl;
	GBS[k * (Nxl + 2) + i].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GBS[k * (Nxl + 2) + i].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GBS[k * (Nxl + 2) + i].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GBS[k * (Nxl + 2) + i].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GBS[k * (Nxl + 2) + i].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
	j = int((thread - k * (Nxl + 2) * (Nyl + 2)) / Nxl);
	i = 1;
	GLS[k * (Nyl + 2) + j].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GLS[k * (Nyl + 2) + j].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GLS[k * (Nyl + 2) + j].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GLS[k * (Nyl + 2) + j].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GLS[k * (Nyl + 2) + j].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
	i = Nxl;
	GRS[k * (Nyl + 2) + j].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GRS[k * (Nyl + 2) + j].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GRS[k * (Nyl + 2) + j].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GRS[k * (Nyl + 2) + j].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GRS[k * (Nyl + 2) + j].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
	i = int(thread - k * (Nxl + 2) * (Nyl + 2) - Nxl * j);
	k = 1;
	GDS[j * (Nxl + 2) + i].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GDS[j * (Nxl + 2) + i].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GDS[j * (Nxl + 2) + i].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GDS[j * (Nxl + 2) + i].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GDS[j * (Nxl + 2) + i].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
	k = Nzl;
	GUS[j * (Nxl + 2) + i].x = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x;
	GUS[j * (Nxl + 2) + i].y = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y;
	GUS[j * (Nxl + 2) + i].z = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z;
	GUS[j * (Nxl + 2) + i].oldvalue = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue;
	GUS[j * (Nxl + 2) + i].value = GCP[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value;
}

void cpuf(int Nxl, int Nyl, int Nzl, Point* CenterPoints, float tauh)
{
	float value;
	for (int k = 1; k < Nzl + 1; k += 1)
	{
		for (int j = 1; j < Nyl + 1; j += 1)
		{
			for (int i = 1; i < Nxl + 1; i += 1)
			{
				Point current = CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i];
				value = current.value;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = 2 * value - current.oldvalue + tauh * (CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i - 1].value + CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i + 1].value + CenterPoints[k * (Nxl + 2) * (Nyl + 2) + (j - 1) * (Nxl + 2) + i].value + CenterPoints[k * (Nxl + 2) * (Nyl + 2) + (j + 1) * (Nxl + 2) + i].value + CenterPoints[(k - 1) * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value + CenterPoints[(k + 1) * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value - 6 * value);
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
			}
		}
	}
}

void Initvalues(int Nxl, int Nyl, int Nzl, Point* CenterPoints, int px, int py, int pz, float tauh2, float h, float Lx, float Ly, float Lz, int n)
{
	float x, y, z, value;
	int i, j, k;
	for (int k = 0; k < Nzl + 2; k += 1)
	{
		for (int j = 0; j < Nyl + 2; j += 1)
		{
			for (int i = 0; i < Nxl + 2; i += 1)
			{
				z = pz * Nzl * h + k * h - h;
				y = py * Nyl * h + j * h - h;
				x = px * Nxl * h + i * h - h;
				value = phi(Lx, Ly, Lz, x, y, z);
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = value + tauh2 * (phi(Lx, Ly, Lz, x - h, y, z) + phi(Lx, Ly, Lz, x + h, y, z) + phi(Lx, Ly, Lz, x, y - h, z) + phi(Lx, Ly, Lz, x, y + h, z) + phi(Lx, Ly, Lz, x, y, z - h) + phi(Lx, Ly, Lz, x, y, z + h) - 6 * value);
			}
		}
	}

	float xmin, ymin, zmin, xmax, ymax, zmax;

	xmin = CenterPoints[0].x;
	xmax = CenterPoints[n - 1].x;
	ymin = CenterPoints[0].y;
	ymax = CenterPoints[n - 1].y;
	zmin = CenterPoints[0].z;
	zmax = CenterPoints[n - 1].z;
	if (ymin < 0)
	{
		j = 0;
		y = ymin;
		for (int k = 0; k < Nzl + 2; k += 1)
		{
			for (int i = 0; i < Nxl + 2; i += 1)
			{
				z = zmin + k * h;
				x = xmin + i * h;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = 0;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = 0;
			}
		}
	}
	else
	{
		j = 0;
		y = ymin;
		for (int k = 0; k < Nzl + 2; k += 1)
		{
			for (int i = 0; i < Nxl + 2; i += 1)
			{
				z = zmin + k * h;
				x = xmin + i * h;
				value = phi(Lx, Ly, Lz, x, y, z);
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = value + tauh2 * (phi(Lx, Ly, Lz, x - h, y, z) + phi(Lx, Ly, Lz, x + h, y, z) + phi(Lx, Ly, Lz, x, y - h, z) + phi(Lx, Ly, Lz, x, y + h, z) + phi(Lx, Ly, Lz, x, y, z - h) + phi(Lx, Ly, Lz, x, y, z + h) - 6 * value);
			}
		}
	}

	if (ymax > Lx)
	{
		j = Nyl + 1;
		y = ymax;
		for (int k = 0; k < Nzl + 2; k += 1)
		{
			for (int i = 0; i < Nxl + 2; i += 1)
			{
				z = zmin + k * h;
				x = xmin + i * h;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = 0;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = 0;
			}
		}
	}
	else
	{
		j = Nyl + 1;
		y = ymax;
		for (int k = 0; k < Nzl + 2; k += 1)
		{
			for (int i = 0; i < Nxl + 2; i += 1)
			{
				z = zmin + k * h;
				x = xmin + i * h;
				value = phi(Lx, Ly, Lz, x, y, z);
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = value + tauh2 * (phi(Lx, Ly, Lz, x - h, y, z) + phi(Lx, Ly, Lz, x + h, y, z) + phi(Lx, Ly, Lz, x, y - h, z) + phi(Lx, Ly, Lz, x, y + h, z) + phi(Lx, Ly, Lz, x, y, z - h) + phi(Lx, Ly, Lz, x, y, z + h) - 6 * value);
			}
		}
	}

	if (zmin < 0)
	{
		k = 0;
		z = zmin;
		for (int j = 0; j < Nyl + 2; j += 1)
		{
			for (int i = 0; i < Nxl + 2; i += 1)
			{
				y = ymin + j * h;
				x = xmin + i * h;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = 0;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = 0;
			}
		}
	}
	else
	{
		k = 0;
		z = zmin;
		for (int j = 0; j < Nyl + 2; j += 1)
		{
			for (int i = 0; i < Nxl + 2; i += 1)
			{
				y = ymin + j * h;
				x = xmin + i * h;
				value = phi(Lx, Ly, Lz, x, y, z);
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = value + tauh2 * (phi(Lx, Ly, Lz, x - h, y, z) + phi(Lx, Ly, Lz, x + h, y, z) + phi(Lx, Ly, Lz, x, y - h, z) + phi(Lx, Ly, Lz, x, y + h, z) + phi(Lx, Ly, Lz, x, y, z - h) + phi(Lx, Ly, Lz, x, y, z + h) - 6 * value);
			}
		}
	}

	if (zmax > Lz)
	{
		k = Nzl + 1;
		z = zmax;
		for (int j = 0; j < Nyl + 2; j += 1)
		{
			for (int i = 0; i < Nxl + 2; i += 1)
			{
				y = ymin + j * h;
				x = xmin + i * h;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = 0;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = 0;
			}
		}
	}
	else
	{
		k = Nzl + 1;
		z = zmax;
		for (int j = 0; j < Nyl + 2; j += 1)
		{
			for (int i = 0; i < Nxl + 2; i += 1)
			{
				y = ymin + j * h;
				x = xmin + i * h;
				value = phi(Lx, Ly, Lz, x, y, z);
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = value + tauh2 * (phi(Lx, Ly, Lz, x - h, y, z) + phi(Lx, Ly, Lz, x + h, y, z) + phi(Lx, Ly, Lz, x, y - h, z) + phi(Lx, Ly, Lz, x, y + h, z) + phi(Lx, Ly, Lz, x, y, z - h) + phi(Lx, Ly, Lz, x, y, z + h) - 6 * value);
			}
		}
	}

	if (xmin < 0)
	{
		i = 0;
		x = Lx;
		for (int k = 0; k < Nzl + 2; k += 1)
		{
			for (int j = 0; j < Nyl + 2; j += 1)
			{
				z = zmin + k * h;
				y = ymin + j * h;
				value = phi(Lx, Ly, Lz, x, y, z);
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = value + tauh2 * (phi(Lx, Ly, Lz, x - h, y, z) + phi(Lx, Ly, Lz, x + h, y, z) + phi(Lx, Ly, Lz, x, y - h, z) + phi(Lx, Ly, Lz, x, y + h, z) + phi(Lx, Ly, Lz, x, y, z - h) + phi(Lx, Ly, Lz, x, y, z + h) - 6 * value);
			}
		}
	}
	else
	{
		i = 0;
		x = xmin;
		for (int k = 0; k < Nzl + 2; k += 1)
		{
			for (int j = 0; j < Nyl + 2; j += 1)
			{
				z = zmin + k * h;
				y = ymin + j * h;
				value = phi(Lx, Ly, Lz, x, y, z);
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = value + tauh2 * (phi(Lx, Ly, Lz, x - h, y, z) + phi(Lx, Ly, Lz, x + h, y, z) + phi(Lx, Ly, Lz, x, y - h, z) + phi(Lx, Ly, Lz, x, y + h, z) + phi(Lx, Ly, Lz, x, y, z - h) + phi(Lx, Ly, Lz, x, y, z + h) - 6 * value);
			}
		}
	}

	if (xmax > Lx)
	{
		i = Nxl + 1;
		x = 0;
		for (int k = 0; k < Nzl + 2; k += 1)
		{
			for (int j = 0; j < Nyl + 2; j += 1)
			{
				z = zmin + k * h;
				y = ymin + j * h;
				value = phi(Lx, Ly, Lz, x, y, z);
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = value + tauh2 * (phi(Lx, Ly, Lz, x - h, y, z) + phi(Lx, Ly, Lz, x + h, y, z) + phi(Lx, Ly, Lz, x, y - h, z) + phi(Lx, Ly, Lz, x, y + h, z) + phi(Lx, Ly, Lz, x, y, z - h) + phi(Lx, Ly, Lz, x, y, z + h) - 6 * value);
			}
		}
	}
	else
	{
		i = 0;
		x = xmax;
		for (int k = 0; k < Nzl + 2; k += 1)
		{
			for (int j = 0; j < Nyl + 2; j += 1)
			{
				z = zmin + k * h;
				y = ymin + j * h;
				value = phi(Lx, Ly, Lz, x, y, z);
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].x = x;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].y = y;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].z = z;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].oldvalue = value;
				CenterPoints[k * (Nxl + 2) * (Nyl + 2) + j * (Nxl + 2) + i].value = value + tauh2 * (phi(Lx, Ly, Lz, x - h, y, z) + phi(Lx, Ly, Lz, x + h, y, z) + phi(Lx, Ly, Lz, x, y - h, z) + phi(Lx, Ly, Lz, x, y + h, z) + phi(Lx, Ly, Lz, x, y, z - h) + phi(Lx, Ly, Lz, x, y, z + h) - 6 * value);
			}
		}
	}
}

void SendRecvValues(int Nxl, int Nyl, int Nzl, int px, int py, int pz, int rx, int ry, int rz, int Np, Point* DownSend, Point* UpSend, Point* LeftSend, Point* RightSend, Point* BehindSend, Point* FrontSend, Point* DownRecv, Point* UpRecv, Point* LeftRecv, Point* RightRecv, Point* BehindRecv, Point* FrontRecv)
{
	MPI_Status status;
	MPI_Datatype dt_point;
	MPI_Type_contiguous(5, MPI_FLOAT, &dt_point);
	MPI_Type_commit(&dt_point);
	int rank, size;
	MPI_Comm_size(MPI_COMM_WORLD, &size);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	int UpComp, DownComp, LeftComp, RightComp, FrontComp, BehindComp;

	UpComp = rx * ry * (pz + 1) + py * rx + px;
	DownComp = rx * ry * (pz - 1) + py * rx + px;

	LeftComp = rx * ry * pz + py * rx + px - 1;
	RightComp = rx * ry * pz + py * rx + px + 1;

	FrontComp = rx * ry * pz + (py - 1) * rx + px;
	BehindComp = rx * ry * pz + (py + 1) * rx + px;

	MPI_Request requests1[2], requests2[2], requests3[2], requests4[2], requests5[2], requests6[2];
	MPI_Status statuses1[2], statuses2[2], statuses3[2], statuses4[2], statuses5[2], statuses6[2];

	if ((0 <= UpComp) && (UpComp < Np))
	{
		MPI_Isend(&UpSend[0], (Nxl + 2) * (Nyl + 2), dt_point, UpComp, 1, MPI_COMM_WORLD, &requests1[0]);

		MPI_Irecv(&UpRecv[0], (Nxl + 2) * (Nyl + 2), dt_point, UpComp, 1, MPI_COMM_WORLD, &requests1[1]);
	}

	if ((0 <= DownComp) && (DownComp < Np))
	{
		MPI_Isend(&DownSend[0], (Nxl + 2) * (Nyl + 2), dt_point, DownComp, 1, MPI_COMM_WORLD, &requests2[0]);

		MPI_Irecv(&DownRecv[0], (Nxl + 2) * (Nyl + 2), dt_point, DownComp, 1, MPI_COMM_WORLD, &requests2[1]);
	}

	if ((0 <= LeftComp) && (LeftComp < Np))
	{
		MPI_Isend(&LeftSend[0], (Nyl + 2) * (Nzl + 2), dt_point, LeftComp, 1, MPI_COMM_WORLD, &requests3[0]);

		MPI_Irecv(&LeftRecv[0], (Nyl + 2) * (Nzl + 2), dt_point, LeftComp, 1, MPI_COMM_WORLD, &requests3[1]);
	}
	else
	{
		LeftComp += rx;
		MPI_Isend(&LeftSend[0], (Nyl + 2) * (Nzl + 2), dt_point, LeftComp, 1, MPI_COMM_WORLD, &requests3[0]);

		MPI_Irecv(&LeftRecv[0], (Nyl + 2) * (Nzl + 2), dt_point, LeftComp, 1, MPI_COMM_WORLD, &requests3[1]);
	}

	if ((0 <= RightComp) && (RightComp < Np))
	{
		MPI_Isend(&RightSend[0], (Nyl + 2) * (Nzl + 2), dt_point, RightComp, 1, MPI_COMM_WORLD, &requests4[0]);

		MPI_Irecv(&RightRecv[0], (Nyl + 2) * (Nzl + 2), dt_point, RightComp, 1, MPI_COMM_WORLD, &requests4[1]);
	}
	else
	{
		RightComp -= rx;
		MPI_Isend(&RightSend[0], (Nyl + 2) * (Nzl + 2), dt_point, RightComp, 1, MPI_COMM_WORLD, &requests4[0]);

		MPI_Irecv(&RightRecv[0], (Nyl + 2) * (Nzl + 2), dt_point, RightComp, 1, MPI_COMM_WORLD, &requests4[1]);
	}

	if ((0 <= FrontComp) && (FrontComp < Np))
	{
		MPI_Isend(&FrontSend[0], (Nxl + 2) * (Nzl + 2), dt_point, FrontComp, 1, MPI_COMM_WORLD, &requests5[0]);

		MPI_Irecv(&FrontRecv[0], (Nxl + 2) * (Nzl + 2), dt_point, FrontComp, 1, MPI_COMM_WORLD, &requests5[1]);
	}

	if ((0 <= BehindComp) && (BehindComp < Np))
	{
		MPI_Isend(&BehindSend[0], (Nxl + 2) * (Nzl + 2), dt_point, BehindComp, 1, MPI_COMM_WORLD, &requests6[0]);

		MPI_Irecv(&BehindRecv[0], (Nxl + 2) * (Nzl + 2), dt_point, BehindComp, 1, MPI_COMM_WORLD, &requests6[1]);
	}

	MPI_Barrier(MPI_COMM_WORLD);
}


void show(Point* P, int n)
{
	for (int i = 0; i < n; i += 1)
	{
		cout << P[i].x << "  " << P[i].y << "  " << P[i].z << "  " << P[i].value << "  " << endl;
	}
}

int main(int argc, char* argv[])
{
	int rank, size;
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &size);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Status status;

	float Lx = 1;
	float Ly = 1;
	float Lz = 1;
	float at = sqrt(1 / (Lx * Lx) + 4 / (Lx * Lx) + 9 / (Lz * Lz)) * Pi;
	float tau = 0.000001;
	int P = 512;
	int Nx = P;
	int Ny = P;
	int Nz = P;
	float h = Lx / (Nx - 1);
	float tauh = tau * tau / (h * h);
	float tauh2 = tau * tau / (2 * h * h);
	int Np = size;

	int rx, ry, rz;
	multi(Np, rx, ry, rz);

	int pz = int(rank / (rx * ry));
	int py = int((rank - pz * rx * ry) / rx);
	int px = int(rank - pz * rx * ry - rx * py);

	int Nxl = Nx / rx;
	int Nyl = Ny / ry;
	int Nzl = Nz / rz;

	int n = (Nxl + 2) * (Nyl + 2) * (Nzl + 2);

	Point* CenterPoints = new Point[n];
	Point* DownSend = new Point[(Nxl + 2) * (Nyl + 2)];
	Point* UpSend = new Point[(Nxl + 2) * (Nyl + 2)];
	Point* LeftSend = new Point[(Nyl + 2) * (Nzl + 2)];
	Point* RightSend = new Point[(Nyl + 2) * (Nzl + 2)];
	Point* FrontSend = new Point[(Nxl + 2) * (Nzl + 2)];
	Point* BehindSend = new Point[(Nxl + 2) * (Nzl + 2)];

	Point* DownRecv = new Point[(Nxl + 2) * (Nyl + 2)];
	Point* UpRecv = new Point[(Nxl + 2) * (Nyl + 2)];
	Point* LeftRecv = new Point[(Nyl + 2) * (Nzl + 2)];
	Point* RightRecv = new Point[(Nyl + 2) * (Nzl + 2)];
	Point* FrontRecv = new Point[(Nxl + 2) * (Nzl + 2)];
	Point* BehindRecv = new Point[(Nxl + 2) * (Nzl + 2)];

	Point* GCP;

	Point* GDS;
	Point* GUS;
	Point* GLS;
	Point* GRS;
	Point* GFS;
	Point* GBS;

	Point* GDR;
	Point* GUR;
	Point* GLR;
	Point* GRR;
	Point* GFR;
	Point* GBR;
	float* error;


	int gsize = n * 5 * sizeof(float);
	int gsizfb = (Nxl + 2) * (Nzl + 2) * 5 * sizeof(float);
	int gsizlr = (Nyl + 2) * (Nzl + 2) * 5 * sizeof(float);
	int gsizud = (Nxl + 2) * (Nyl + 2) * 5 * sizeof(float);


	const int blocksize = 512;
	int gridsize = (n - 1) / blocksize + 1;

	double start;
	if (rank == 0)
	{
		start = MPI_Wtime();
	}

	Initvalues(Nxl, Nyl, Nzl, CenterPoints, px, py, pz, tauh2, h, Lx, Ly, Lz, n);

	hipMalloc((void**)&GCP, gsize);
	hipMemcpy(GCP, CenterPoints, gsize, hipMemcpyHostToDevice);

	hipMalloc((void**)&GDS, gsizud);
	hipMalloc((void**)&GUS, gsizud);
	hipMalloc((void**)&GLS, gsizlr);
	hipMalloc((void**)&GRS, gsizlr);
	hipMalloc((void**)&GFS, gsizfb);
	hipMalloc((void**)&GBS, gsizfb);

	hipMalloc((void**)&GDR, gsizud);
	hipMalloc((void**)&GUR, gsizud);
	hipMalloc((void**)&GLR, gsizlr);
	hipMalloc((void**)&GRR, gsizlr);
	hipMalloc((void**)&GFR, gsizfb);
	hipMalloc((void**)&GBR, gsizfb);
	float errorCPU;
	float* errorGPU;
	hipMalloc((void**)&errorGPU, sizeof(float));

	Kernel1 << < gridsize, blocksize >> > (Nxl, Nyl, Nzl, GCP, GLS, GRS, GDS, GUS, GFS, GBS, tauh);

	hipMemcpy(LeftSend, GLS, gsizlr, hipMemcpyDeviceToHost);
	hipMemcpy(RightSend, GRS, gsizlr, hipMemcpyDeviceToHost);
	hipMemcpy(UpSend, GUS, gsizud, hipMemcpyDeviceToHost);
	hipMemcpy(DownSend, GDS, gsizud, hipMemcpyDeviceToHost);
	hipMemcpy(BehindSend, GBS, gsizfb, hipMemcpyDeviceToHost);
	hipMemcpy(FrontSend, GFS, gsizfb, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	SendRecvValues(Nxl, Nyl, Nzl, px, py, pz, rx, ry, rz, Np, DownSend, UpSend, LeftSend, RightSend, BehindSend, FrontSend, DownRecv, UpRecv, LeftRecv, RightRecv, BehindRecv, FrontRecv);

	float ALLERROR;

	for (int t1 = 1; t1 < 20; t1 += 1)
	{

		hipMemcpy(GLR, LeftRecv, gsizlr, hipMemcpyHostToDevice);
		hipMemcpy(GRR, RightRecv, gsizlr, hipMemcpyHostToDevice);
		hipMemcpy(GUR, UpRecv, gsizud, hipMemcpyHostToDevice);
		hipMemcpy(GDR, DownRecv, gsizud, hipMemcpyHostToDevice);
		hipMemcpy(GBR, BehindRecv, gsizfb, hipMemcpyHostToDevice);
		hipMemcpy(GFR, FrontRecv, gsizfb, hipMemcpyHostToDevice);

		Kernel2 << < gridsize, blocksize >> > (Nxl, Nyl, Nzl, GCP, GLS, GRS, GDS, GUS, GFS, GBS, GLR, GRR, GDR, GUR, GFR, GBR, tauh);

		counterror << < 1, blocksize >> > (errorGPU, Nxl, Nyl, Nzl, GCP, tauh, at, Lx, Ly, Lz, t1);

		hipMemcpy(&errorCPU, errorGPU, sizeof(float), hipMemcpyDeviceToHost);

		hipDeviceSynchronize();

		MPI_Reduce(&errorCPU, &ALLERROR, 1, MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);

		if (rank == 0)
		{
			cout << "Слой номер " << t1 << ", ошибка: " << ALLERROR * t1 << endl;
		}

		hipMemcpy(LeftSend, GLS, gsizlr, hipMemcpyDeviceToHost);
		hipMemcpy(RightSend, GRS, gsizlr, hipMemcpyDeviceToHost);
		hipMemcpy(UpSend, GUS, gsizud, hipMemcpyDeviceToHost);
		hipMemcpy(DownSend, GDS, gsizud, hipMemcpyDeviceToHost);
		hipMemcpy(BehindSend, GBS, gsizfb, hipMemcpyDeviceToHost);
		hipMemcpy(FrontSend, GFS, gsizfb, hipMemcpyDeviceToHost);


		SendRecvValues(Nxl, Nyl, Nzl, px, py, pz, rx, ry, rz, Np, DownSend, UpSend, LeftSend, RightSend, BehindSend, FrontSend, DownRecv, UpRecv, LeftRecv, RightRecv, BehindRecv, FrontRecv);

	}
	if (rank == 0)
	{
		double end = MPI_Wtime();
		printf("Time = %3.10f", end - start);
	}



	//hipMemcpy(CenterPoints, GCP, gsize, hipMemcpyDeviceToHost);
	//show(CenterPoints, n);

	hipFree(GDS);
	hipFree(GCP);
	hipFree(GBR);
	hipFree(GBS);
	hipFree(GDR);
	hipFree(GFR);
	hipFree(GFS);
	hipFree(GLR);
	hipFree(GLS);
	hipFree(GRR);
	hipFree(GRS);
	hipFree(GUR);
	hipFree(GUS);
	hipFree(error);

	hipProfilerStop();
	MPI_Finalize();
}
